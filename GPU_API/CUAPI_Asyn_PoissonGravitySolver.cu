#include "hip/hip_runtime.h"
#include "GAMER.h"
#include "CUPOT.h"

#if   ( POT_SCHEME == SOR )
#ifdef USE_PSOLVER_10TO14
__global__ void CUPOT_PoissonSolver_SOR_10to14cube( const real g_Rho_Array    [][ RHO_NXT*RHO_NXT*RHO_NXT ], 
                                                    const real g_Pot_Array_In [][ POT_NXT*POT_NXT*POT_NXT ], 
                                                          real g_Pot_Array_Out[][ GRA_NXT*GRA_NXT*GRA_NXT ],
                                                    const int Min_Iter, const int Max_Iter, const real Omega_6,
                                                    const real Const, const IntScheme_t IntScheme );
#else
__global__ void CUPOT_PoissonSolver_SOR_16to18cube( const real g_Rho_Array    [][ RHO_NXT*RHO_NXT*RHO_NXT ], 
                                                    const real g_Pot_Array_In [][ POT_NXT*POT_NXT*POT_NXT ], 
                                                          real g_Pot_Array_Out[][ GRA_NXT*GRA_NXT*GRA_NXT ],
                                                    const int Min_Iter, const int Max_Iter, const real Omega_6, 
                                                    const real Const, const IntScheme_t IntScheme );
#endif // #ifdef USE_PSOLVER_10TO14 ... else ...
#elif ( POT_SCHEME == MG  )
__global__ void CUPOT_PoissonSolver_MG( const real g_Rho_Array    [][ RHO_NXT*RHO_NXT*RHO_NXT ], 
                                        const real g_Pot_Array_In [][ POT_NXT*POT_NXT*POT_NXT ], 
                                              real g_Pot_Array_Out[][ GRA_NXT*GRA_NXT*GRA_NXT ],
                                        const real dh_Min, const int Max_Iter, const int NPre_Smooth,
                                        const int NPost_Smooth, const real Tolerated_Error, const real Poi_Coeff,
                                        const IntScheme_t IntScheme );
#endif // POT_SCHEME
__global__ void CUPOT_GravitySolver(       real g_Flu_Array[][5][ PATCH_SIZE*PATCH_SIZE*PATCH_SIZE ],
                                     const real g_Pot_Array[][ GRA_NXT*GRA_NXT*GRA_NXT ],
                                     const real Gra_Const, const bool P5_Gradient );


// declare all device pointers
real (*d_Rho_Array_P    )[ RHO_NXT*RHO_NXT*RHO_NXT ]                 = NULL;
real (*d_Pot_Array_P_In )[ POT_NXT*POT_NXT*POT_NXT ]                 = NULL;
real (*d_Pot_Array_P_Out)[ GRA_NXT*GRA_NXT*GRA_NXT ]                 = NULL;
real (*d_Flu_Array_G    )[NCOMP][ PATCH_SIZE*PATCH_SIZE*PATCH_SIZE ] = NULL;

// REPLACE in the actual implementation
// #########################################################################
hipStream_t *Stream = NULL;
// extern hipStream_t *Stream;
// #########################################################################




//-------------------------------------------------------------------------------------------------------
// Function    :  CUAPI_Asyn_PoissonGravitySolver
// Description :  Invoke the CUPOT_PoissonSolver_XXtoXXcube and/or CUPOT_GravitySolver kernel(s) to evaluate 
//                the gravitational potential and/or advance the fluid variables by the gravitational
//                acceleration for a group of patches
//
//                ***********************************************************
//                **                Asynchronous Function                  **
//                **                                                       ** 
//                **  will return before the execution in GPU is complete  **
//                ***********************************************************
//
// Note        :  a. Use streams for the asychronous memory copy between device and host
//                b. Prefix "d" : for pointers pointing to the "Device" memory space
//                   Prefix "h" : for pointers pointing to the "Host"   memory space
//
// Parameter   :  h_Rho_Array          : Host array to store the input density 
//                h_Pot_Array_In       : Host array to store the input "coarse-grid" potential for interpolation
//                h_Pot_Array_Out      : Host array to store the output potential
//                h_Flu_Array          : Host array to store the fluid variables for the Gravity solver
//                NPatchGroup          : Number of patch groups evaluated simultaneously by GPU 
//                dt                   : Time interval to advance solution
//                dh                   : Grid size
//                SOR_Min_Iter         : Minimum # of iterations for SOR
//                SOR_Max_Iter         : Maximum # of iterations for SOR
//                SOR_Omega            : Over-relaxation parameter
//                MG_Max_Iter          : Maximum number of iterations for multigrid
//                MG_NPre_Smooth       : Number of pre-smoothing steps for multigrid
//                MG_NPos_tSmooth      : Number of post-smoothing steps for multigrid
//                MG_Tolerated_Error   : Maximum tolerated error for multigrid
//                Poi_Coeff            : Coefficient in front of density in the Poisson equation (4*Pi*Newton_G*a)
//                IntScheme            : Interpolation scheme for potential : 
//                                       4 --> conservative quadratic interpolation
//                                       5 --> quadratic interpolation
//                P5_Gradient          : Use 5-points stencil to evaluate the potential gradient
//                Poisson              : true --> invoke the Poisson solver
//                GraAcc               : true --> invoke the Gravity solver
//                GPU_NStream          : Number of CUDA streams for the asynchronous memory copy
//-------------------------------------------------------------------------------------------------------
void CUAPI_Asyn_PoissonGravitySolver( const real h_Rho_Array    [][RHO_NXT][RHO_NXT][RHO_NXT], 
                                            real h_Pot_Array_In [][POT_NXT][POT_NXT][POT_NXT],
                                            real h_Pot_Array_Out[][GRA_NXT][GRA_NXT][GRA_NXT],
                                            real h_Flu_Array    [][5][PATCH_SIZE][PATCH_SIZE][PATCH_SIZE], 
                                      const int NPatchGroup, const real dt, const real dh, const int SOR_Min_Iter,
                                      const int SOR_Max_Iter, const real SOR_Omega, const int MG_Max_Iter,
                                      const int MG_NPre_Smooth, const int MG_NPost_Smooth, 
                                      const real MG_Tolerated_Error, const real Poi_Coeff,
                                      const IntScheme_t IntScheme, const bool P5_Gradient, const bool Poisson, 
                                      const bool GraAcc, const int GPU_NStream )
{

#  if   ( POT_SCHEME == SOR )
   const dim3 Poi_Block_Dim( RHO_NXT/2, RHO_NXT, POT_BLOCK_SIZE_Z );
#  elif ( POT_SCHEME == MG )
   const dim3 Poi_Block_Dim( POT_BLOCK_SIZE_X, 1, 1 );
#  endif
   const dim3 Gra_Block_Dim( PATCH_SIZE, PATCH_SIZE, GRA_BLOCK_SIZE_Z );
   const int  NPatch      = NPatchGroup*8;
   const int  Poi_NThread = Poi_Block_Dim.x * Poi_Block_Dim.y * Poi_Block_Dim.z;
#  if   ( POT_SCHEME == SOR )
   const real Poi_Const   = Poi_Coeff*dh*dh;
   const real SOR_Omega_6 = SOR_Omega/6.0;
#  endif

   real Gra_Const;

   if ( P5_Gradient )   Gra_Const = dt/(12.0*dh);
   else                 Gra_Const = dt/( 2.0*dh);


// minimum number of threads for spatial interpolation
   if ( Poi_NThread < (POT_NXT-2)*(POT_NXT-2) )
      Aux_Error( ERROR_INFO, "Poi_NThread (%d) < (POT_NXT-2)*(POT_NXT-2) (%d) !!\n", 
                 Poi_NThread, (POT_NXT-2)*(POT_NXT-2) );

// constraint due to the reduction operation in "CUPOT_Poisson_10to14cube" and "CUPOT_PoissonSolver_MG"
#  if (  ( POT_SCHEME == SOR && defined USE_PSOLVER_10TO14 )  ||  POT_SCHEME == MG  )
   if ( Poi_NThread < 64 )
      Aux_Error( ERROR_INFO, "incorrect parameter %s = %d (must >= 64) !!\n", "Poi_NThread", Poi_NThread );
#  endif

// constraint in "CUPOT_PoissonSolver_SOR_16to18cube"
#  if ( POT_SCHEME == SOR  &&  !defined USE_PSOLVER_10TO14 )
   if ( Poi_NThread != RHO_NXT*RHO_NXT/2 )
      Aux_Error( ERROR_INFO, "incorrect parameter %s = %d (must == %d) !!\n", "Poi_NThread", Poi_NThread,
                 RHO_NXT*RHO_NXT/2 );
#  endif

   if ( IntScheme != INT_CQUAD  &&  IntScheme != INT_QUAD )
      Aux_Error( ERROR_INFO, "incorrect parameter %s = %d !!\n", "IntScheme", IntScheme );

#  if ( GRA_GHOST_SIZE == 1 )
   if ( P5_Gradient )
      Aux_Error( ERROR_INFO, "incorrect parameter %s = %d !!\n", "PT_Gradient", P5_Gradient );
#  endif


   int *NPatch_per_Stream = new int [GPU_NStream];
   int *Rho_MemSize       = new int [GPU_NStream];
   int *Pot_MemSize_In    = new int [GPU_NStream];
   int *Pot_MemSize_Out   = new int [GPU_NStream];
   int *Flu_MemSize       = new int [GPU_NStream];
   int *UsedPatch         = new int [GPU_NStream];


// set the number of patches in each stream
   UsedPatch[0] = 0;

   if ( GPU_NStream == 1 )    NPatch_per_Stream[0] = NPatch;
   else
   {
      for (int s=0; s<GPU_NStream-1; s++)    
      {
         NPatch_per_Stream[s] = NPatch/GPU_NStream;
         UsedPatch[s+1] = UsedPatch[s] + NPatch_per_Stream[s];
      }

      NPatch_per_Stream[GPU_NStream-1] = NPatch - UsedPatch[GPU_NStream-1];
   }


// set the size of data to be transferred into GPU in each stream
   for (int s=0; s<GPU_NStream; s++)
   {
      Rho_MemSize    [s] = NPatch_per_Stream[s]*RHO_NXT   *RHO_NXT   *RHO_NXT   *sizeof(real);
      Pot_MemSize_In [s] = NPatch_per_Stream[s]*POT_NXT   *POT_NXT   *POT_NXT   *sizeof(real);
      Pot_MemSize_Out[s] = NPatch_per_Stream[s]*GRA_NXT   *GRA_NXT   *GRA_NXT   *sizeof(real);
      Flu_MemSize    [s] = NPatch_per_Stream[s]*PATCH_SIZE*PATCH_SIZE*PATCH_SIZE*sizeof(real)*NCOMP;
   }


// a. copy data from host to device
//=========================================================================================
   for (int s=0; s<GPU_NStream; s++)
   {
      if ( NPatch_per_Stream[s] == 0 )    continue;

      if ( Poisson )
      {
         CUDA_CHECK_ERROR(  hipMemcpyAsync( d_Rho_Array_P     + UsedPatch[s], h_Rho_Array     + UsedPatch[s], 
                                             Rho_MemSize[s],     hipMemcpyHostToDevice, Stream[s] )  );

         CUDA_CHECK_ERROR(  hipMemcpyAsync( d_Pot_Array_P_In  + UsedPatch[s], h_Pot_Array_In  + UsedPatch[s],
                                             Pot_MemSize_In[s],  hipMemcpyHostToDevice, Stream[s] )  );
      }

      if ( GraAcc )
      {
         if ( !Poisson )
         CUDA_CHECK_ERROR(  hipMemcpyAsync( d_Pot_Array_P_Out + UsedPatch[s], h_Pot_Array_Out + UsedPatch[s],
                                             Pot_MemSize_Out[s], hipMemcpyHostToDevice, Stream[s] )  );

         CUDA_CHECK_ERROR(  hipMemcpyAsync( d_Flu_Array_G     + UsedPatch[s], h_Flu_Array     + UsedPatch[s], 
                                             Flu_MemSize[s],     hipMemcpyHostToDevice, Stream[s] )  );
      }
   } // for (int s=0; s<GPU_NStream; s++)


// b. execute the kernel 
//=========================================================================================
   for (int s=0; s<GPU_NStream; s++)
   {
      if ( NPatch_per_Stream[s] == 0 )    continue;

      if ( Poisson )
      {
#        if ( POT_SCHEME == SOR )

#        ifdef USE_PSOLVER_10TO14
         CUPOT_PoissonSolver_SOR_10to14cube <<< NPatch_per_Stream[s], Poi_Block_Dim, 0, Stream[s] >>> 
                                            ( d_Rho_Array_P + UsedPatch[s], d_Pot_Array_P_In + UsedPatch[s], 
                                              d_Pot_Array_P_Out + UsedPatch[s], SOR_Min_Iter, SOR_Max_Iter, 
                                              SOR_Omega_6, Poi_Const, IntScheme );
#        else
         CUPOT_PoissonSolver_SOR_16to18cube <<< NPatch_per_Stream[s], Poi_Block_Dim, 0, Stream[s] >>> 
                                            ( d_Rho_Array_P + UsedPatch[s], d_Pot_Array_P_In + UsedPatch[s], 
                                              d_Pot_Array_P_Out + UsedPatch[s], SOR_Min_Iter, SOR_Max_Iter, 
                                              SOR_Omega_6, Poi_Const, IntScheme );
#        endif // #ifdef USE_PSOLVER_10TO14 ... else ...

#        elif ( POT_SCHEME == MG  )

         CUPOT_PoissonSolver_MG             <<< NPatch_per_Stream[s], Poi_Block_Dim, 0, Stream[s] >>> 
                                            ( d_Rho_Array_P + UsedPatch[s], d_Pot_Array_P_In + UsedPatch[s], 
                                              d_Pot_Array_P_Out + UsedPatch[s],
                                              dh, MG_Max_Iter, MG_NPre_Smooth, MG_NPost_Smooth, 
                                              MG_Tolerated_Error, Poi_Coeff, IntScheme );

#        else

         #error : unsupported GPU Poisson solver

#        endif
      } // if ( Poisson )

      if ( GraAcc )
      {
         CUPOT_GravitySolver <<< NPatch_per_Stream[s], Gra_Block_Dim, 0, Stream[s] >>> 
                             ( d_Flu_Array_G + UsedPatch[s], d_Pot_Array_P_Out + UsedPatch[s], Gra_Const, 
                               P5_Gradient );
      }

      CUDA_CHECK_ERROR( hipGetLastError() );
   } // for (int s=0; s<GPU_NStream; s++)


// c. copy data from device to host
//=========================================================================================
   for (int s=0; s<GPU_NStream; s++)
   {
      if ( NPatch_per_Stream[s] == 0 )    continue;

      if ( Poisson )
         CUDA_CHECK_ERROR(  hipMemcpyAsync( h_Pot_Array_Out + UsedPatch[s], d_Pot_Array_P_Out + UsedPatch[s], 
                                             Pot_MemSize_Out[s], hipMemcpyDeviceToHost, Stream[s] )  );

      if ( GraAcc )
         CUDA_CHECK_ERROR(  hipMemcpyAsync( h_Flu_Array     + UsedPatch[s], d_Flu_Array_G     + UsedPatch[s], 
                                             Flu_MemSize[s],     hipMemcpyDeviceToHost, Stream[s] )  );
   } // for (int s=0; s<GPU_NStream; s++)


   delete [] NPatch_per_Stream;
   delete [] Rho_MemSize;
   delete [] Pot_MemSize_In;
   delete [] Pot_MemSize_Out;
   delete [] Flu_MemSize;
   delete [] UsedPatch;

} // FUNCTION : CUAPI_Asyn_PoissonGravitySolver



